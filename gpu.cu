
#include <hip/hip_runtime.h>
//
// Created by Emily & Marc on 10/12/2018.
//

#ifdef CUDA_AVAILABLE

#include "gpu.h"

using namespace utils;

namespace gpu {

    template<unsigned int blockSize, typename kernelHandle>
    float gm_call_kernel(kernelHandle &kernel, const float *U, const float *V, float a, int k, unsigned int n) {
        int numBlocks = (n + blockSize - 1) / blockSize;
        float *sumGpu, *divGpu;
        cudaMallocManaged(&sumGpu, n * sizeof(float));
        cudaMallocManaged(&divGpu, n * sizeof(float));

        kernel << < numBlocks, blockSize, 2 * blockSize * sizeof(float) >> > (U, V, a, k, n, sumGpu, divGpu);
        cudaDeviceSynchronize();

        float sumFinal = 0, divFinal = 0;
        for (unsigned int i = 0; i < numBlocks; i++) {
            sumFinal += sumGpu[i];
            divFinal += divGpu[i];
        }

        cudaFree(sumGpu);
        cudaFree(divGpu);

        return sumFinal / divFinal;
    }

    __device__
    inline float ipowf_helper(float base, int exp) {
        float result = 1;
        while (true) {
            if (exp & 1) result *= base;
            exp >>= 1;
            if (!exp) break;
            base *= base;
        }
        return result;
    }

    __device__
    inline float ipowf(float base, int exp) {
        if (exp > 0) return ipowf_helper(base, exp);
        else if (exp == 0) return 1.f;
        else return 1.f / ipowf_helper(base, -exp);
    }

    __global__
    void gm_kernel(const float *U, const float *V, float a, int k, unsigned int n, float *outSum, float *outDiv) {
        extern __shared__ float sdata[];
        float *sdataSum = sdata;
        float *sdataDiv = sdata + blockDim.x;
        unsigned int tid = threadIdx.x;
        unsigned int index = blockIdx.x * blockDim.x + tid;
        unsigned int stride = blockDim.x * gridDim.x;

        sdataSum[tid] = 0;
        sdataDiv[tid] = 0;

        for (unsigned int i = index; i < n; i += stride) {
            sdataSum[tid] += ipowf(V[i] * U[i] - a, k);
            sdataDiv[tid] += V[i];
        }
        __syncthreads();
        if (tid == 0) {
            outSum[blockIdx.x]=0;
            outDiv[blockIdx.x]=0;
            for (unsigned int i = 0; i < blockDim.x; i++) {
                outSum[blockIdx.x] += sdataSum[i];
                outDiv[blockIdx.x] += sdataDiv[i];
            }
        }
    }

    float gm(const float *U, const float *V, float a, int k, unsigned int n) {
        static const unsigned int blockSize = 256;
        return gm_call_kernel<blockSize>(gm_kernel, U, V, a, k, n);
    }

    template<unsigned int blockSize>
    __device__
    void warpReduce(volatile float *sdata, unsigned int tid) {
        if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
        if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
        if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
        if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
        if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
        if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }

    template<unsigned int blockSize>
    __global__
    void
    gm_kernel_optimized(const float *U, const float *V, float a, int k, unsigned int n, float *outSum, float *outDiv) {
        extern __shared__ float sdata[];
        float *sdataSum = sdata;
        float *sdataDiv = sdata + blockDim.x;
        unsigned int tid = threadIdx.x;
        unsigned int index = blockIdx.x * blockDim.x + tid;
        unsigned int stride = blockDim.x * gridDim.x;

        sdataSum[tid] = 0;
        sdataDiv[tid] = 0;

        for (unsigned int i = index; i < n; i += stride) {
            sdataSum[tid] += ipowf(V[i] * U[i] - a, k);
            sdataDiv[tid] += V[i];
        }
        __syncthreads();

        if (blockSize >= 512) {
            if (tid < 256) {
                sdataSum[tid] += sdataSum[tid + 256];
                sdataDiv[tid] += sdataDiv[tid + 256];
            }
            __syncthreads();
        }
        if (blockSize >= 256) {
            if (tid < 128) {
                sdataSum[tid] += sdataSum[tid + 128];
                sdataDiv[tid] += sdataDiv[tid + 128];
            }
            __syncthreads();
        }
        if (blockSize >= 128) {
            if (tid < 64) {
                sdataSum[tid] += sdataSum[tid + 64];
                sdataDiv[tid] += sdataDiv[tid + 64];
            }
            __syncthreads();
        }

        if (tid < 32) {
            warpReduce<blockSize>(sdataSum, tid);
            warpReduce<blockSize>(sdataDiv, tid);
        }
        if (tid == 0) {
            outSum[blockIdx.x] = sdataSum[0];
            outDiv[blockIdx.x] = sdataDiv[0];
        }
    }


    float gm_optimized(const float *U, const float *V, float a, int k, unsigned int n) {
        static const unsigned int blockSize = 256;
        return gm_call_kernel< blockSize >(gm_kernel_optimized<blockSize>, U, V, a, k, n);
    }

    void initParams(benchmarkParams &params) {
        cudaMalloc(&params.U_gpu, params.n * sizeof(float));
        cudaMalloc(&params.W_gpu, params.n * sizeof(float));
        cudaMemcpy(params.U_gpu, params.U, params.n * sizeof(float), cudaMemcpyHostToDevice);
        cudaMemcpy(params.W_gpu, params.W, params.n * sizeof(float), cudaMemcpyHostToDevice);
        cudaDeviceSynchronize();
    }

    void freeParams(benchmarkParams &params) {
        cudaDeviceSynchronize();
        cudaFree(&params.U_gpu);
        cudaFree(&params.W_gpu);
    }
}
#endif